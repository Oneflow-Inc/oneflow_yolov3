#include "hip/hip_runtime.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void UpsampleNearestForward(const int64_t nthreads, const T* in_dptr,
                                       const int64_t channel_num, const int64_t height,
                                       const int64_t width, const int64_t new_height,
                                       const int64_t new_width, const float scale_h,
                                       const float scale_w, const bool align_corners, T* out_dptr) {
  const int64_t new_area = new_height * new_width;
  const int64_t channel_area = channel_num * height * width;
  const int64_t channel_new_area = channel_num * new_height * new_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / new_width) % new_height;
    const int64_t w = index % new_width;
    const int64_t c = (index / new_area) % channel_num;
    const int64_t n = index / channel_new_area;
    const int64_t in_h = min((align_corners) ? static_cast<int64_t>(roundf(h * scale_h))
                                             : static_cast<int64_t>(floorf(h * scale_h)),
                             height - 1);
    const int64_t in_w = min((align_corners) ? static_cast<int64_t>(roundf(w * scale_w))
                                             : static_cast<int64_t>(floorf(w * scale_w)),
                             width - 1);
    out_dptr[index] = in_dptr[n * channel_area + (c * height + in_h) * width + in_w];
  }
}

template<typename T>
__global__ void UpsampleNearestBackward(const int64_t nthreads, const T* dy_dptr,
                                        const int64_t channel_num, const int64_t height,
                                        const int64_t width, const int64_t new_height,
                                        const int64_t new_width, const float scale_h,
                                        const float scale_w, const bool align_corners, T* dx_dptr) {
  const int64_t area = height * width;
  const int64_t new_area = new_height * new_width;
  const int64_t channel_area = channel_num * height * width;
  const int64_t channel_new_area = channel_num * new_height * new_width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int64_t h = (index / new_width) % new_height;
    const int64_t w = index % new_width;
    const int64_t c = (index / new_area) % channel_num;
    const int64_t n = index / channel_new_area;
    const int64_t in_h = min((align_corners) ? static_cast<int64_t>(roundf(h * scale_h))
                                             : static_cast<int64_t>(floorf(h * scale_h)),
                             height - 1);
    const int64_t in_w = min((align_corners) ? static_cast<int64_t>(roundf(w * scale_w))
                                             : static_cast<int64_t>(floorf(w * scale_w)),
                             width - 1);
    atomicAdd(dx_dptr + n * channel_area + (c * height + in_h) * width + in_w, dy_dptr[index]);
  }
}

}  // namespace

template<typename T>
class UpsampleNearestGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGPUKernel() = default;
  ~UpsampleNearestGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x_blob = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y_blob = ctx->Tensor4ArgNameAndIndex("y", 0);
    const int32_t scale = ctx->Attr<int32_t>("scale");
    const int64_t elem_cnt = y_blob->shape().elem_cnt();
    UpsampleNearestForward<T>
        <<<BlocksNum4ThreadsNum(elem_cnt), 1024, 0, ctx->device_ctx()->hip_stream()>>>(
            elem_cnt, x_blob->dptr<T>(), x_blob->shape().At(1), x_blob->shape().At(2),
            x_blob->shape().At(3), y_blob->shape().At(2), y_blob->shape().At(3), 1.f / scale,
            1.f / scale, false, y_blob->mut_dptr<T>());
  }
    bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }

};

template<typename T>
class UpsampleNearestGradGPUKernel final : public user_op::OpKernel {
 public:
  UpsampleNearestGradGPUKernel() = default;
  ~UpsampleNearestGradGPUKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    user_op::Tensor* dx_blob = ctx->Tensor4ArgNameAndIndex("dx", 0);
    if (dx_blob == nullptr) { return; }
    Memset<DeviceType::kGPU>(ctx->device_ctx(), dx_blob->mut_dptr<T>(), 0,
                             dx_blob->shape().elem_cnt() * sizeof(T));
    const user_op::Tensor* dy_blob = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const int32_t scale = ctx->Attr<int32_t>("scale");
    const int64_t elem_cnt = dy_blob->shape().elem_cnt();
    UpsampleNearestBackward<T>
        <<<BlocksNum4ThreadsNum(elem_cnt), 1024, 0, ctx->device_ctx()->hip_stream()>>>(
            elem_cnt, dy_blob->dptr<T>(), dx_blob->shape().At(1), dx_blob->shape().At(2),
            dx_blob->shape().At(3), dy_blob->shape().At(2), dy_blob->shape().At(3), 1.f / scale,
            1.f / scale, false, dx_blob->mut_dptr<T>());
  }
    bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }

};

#define REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(dtype)                                 \
  REGISTER_USER_KERNEL("upsample_nearest")                                          \
      .SetCreateFn<UpsampleNearestGPUKernel<dtype>>()                               \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) { return true; }); \
  REGISTER_USER_KERNEL("upsample_nearest_grad")                                     \
      .SetCreateFn<UpsampleNearestGPUKernel<dtype>>()                               \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) { return true; });

REGISTER_UPSAMPLE_NEAREST_GPU_KERNEL(float)

REGISTER_USER_OP("upsample_nearest")
    .Input("x")
    .Output("y")
    .Attr("scale", UserOpAttrType::kAtInt32)
    .Attr("data_format", UserOpAttrType::kAtString)
    .SetTensorDescInferFn([](user_op::InferContext* ctx) -> Maybe<void> {
      const Shape* x_shape = ctx->Shape4ArgNameAndIndex("x", 0);
      Shape* y_shape = ctx->Shape4ArgNameAndIndex("y", 0);
      const int32_t scale = ctx->Attr<int32_t>("scale");
      if (ctx->Attr<std::string>("data_format") != "channels_first" || x_shape->NumAxes() != 4) {
        LOG(FATAL) << "upsample_nearest only supports NCHW";
      }
      *y_shape =
          Shape({x_shape->At(0), x_shape->At(1), scale * x_shape->At(2), scale * x_shape->At(3)});
      return Maybe<void>::Ok();
    })
    .SetGetSbpFn([](user_op::SbpContext* ctx) -> Maybe<void> {
      ctx->NewBuilder().Split(user_op::OpArg("x", 0), 0).Split(user_op::OpArg("y", 0), 0).Build();
      return Maybe<void>::Ok();
    });

REGISTER_USER_OP("upsample_nearest_grad")
    .Input("dy")
    .Output("dx")
    .Attr("scale", UserOpAttrType::kAtInt32)
    .Attr("data_format", UserOpAttrType::kAtString)
    .SetTensorDescInferFn([](user_op::InferContext* ctx) -> Maybe<void> {
      const Shape* dy_shape = ctx->Shape4ArgNameAndIndex("dy", 0);
      Shape* dx_shape = ctx->Shape4ArgNameAndIndex("dx", 0);
      const int32_t scale = ctx->Attr<int32_t>("scale");
      if (ctx->Attr<std::string>("data_format") != "channels_first"
          || dy_shape->NumAxes() != 4) {
        LOG(FATAL) << "upsample_nearest only supports NCHW";
      }
      *dx_shape = Shape(
          {dy_shape->At(0), dy_shape->At(1), dy_shape->At(2) / scale, dy_shape->At(3) / scale});
      return Maybe<void>::Ok();
    })
    .SetGetSbpFn([](user_op::SbpContext* ctx) -> Maybe<void> {
      ctx->NewBuilder().Split(user_op::OpArg("dy", 0), 0).Split(user_op::OpArg("dx", 0), 0).Build();
      return Maybe<void>::Ok();
    });

REGISTER_USER_OP_GRAD("upsample_nearest")
    .SetGenBackwardOpConfFn([](const user_op::UserOpWrapper& op, user_op::AddOpFn AddOp) {
      if (op.NeedGenGradTensor4OpInput("x", 0)) {
        user_op::UserOpConfWrapperBuilder builder(op.op_name() + "_grad");
        user_op::UserOpConfWrapper grad_op =
            builder.Op("upsample_nearest_grad")
                .Input("dy", op.GetGradTensorWithOpOutput("y", 0))
                .Output("dx")
                .Attr("scale", op.attr<int32_t>("scale"))
                .Attr("data_format", op.attr<std::string>("data_format"))
                .Build();
        op.BindGradTensorWithOpInput(grad_op.output("dx", 0), "x", 0);
        AddOp(grad_op);
      }
    });

}  // namespace oneflow
